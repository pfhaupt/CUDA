#include "hip/hip_runtime.h"
// #define DEBUG
#define ELEM_TYPE float
#include "common.h"

#include "raylib.h"

#define WIDTH 3840
#define HEIGHT 2160

#define AA_LEVEL 2
#define AA_PER_PIXEL (AA_LEVEL * AA_LEVEL)

#define cast(x, typ) (*((typ*)&(x)))

#define map(val, from, to, low, high) (low + (val - from) / (to - from) * (high - low))
#define lerp(low, high, perc) ((int)(low + (high - low) * perc))

typedef float gpuType;

__device__ static unsigned int colors[79] = {
    0xffff0000,
    0xffff0027,
    0xffff0046,
    0xffff145e,
    0xffff2472,
    0xffff2e85,
    0xffff3597,
    0xffff3aa9,
    0xffff3dbd,
    0xffff3cd3,
    0xffff35ee,
    0xffff34ff,
    0xffe869ff,
    0xffd083ff,
    0xffba93ff,
    0xffa6a0ff,
    0xff92a9ff,
    0xff7eb1ff,
    0xff67b7ff,
    0xff4abcff,
    0xff17c1ff,
    0xff00c5ff,
    0xff00c8ff,
    0xff00caff,
    0xff00ccfb,
    0xff00cef6,
    0xff00d0f2,
    0xff00d3ee,
    0xff00d5e9,
    0xff00d8e3,
    0xff00dbdc,
    0xff00dfd3,
    0xff00e3c9,
    0xff00e7bd,
    0xff00ebaf,
    0xff00ef9f,
    0xff00f48b,
    0xff00f970,
    0xff00fe45,
    0xff00fe45,
    0xff00f970,
    0xff00f48b,
    0xff00ef9f,
    0xff00ebaf,
    0xff00e7bd,
    0xff00e3c9,
    0xff00dfd3,
    0xff00dbdc,
    0xff00d8e3,
    0xff00d5e9,
    0xff00d3ee,
    0xff00d0f2,
    0xff00cef6,
    0xff00ccfb,
    0xff00caff,
    0xff00c8ff,
    0xff00c5ff,
    0xff17c1ff,
    0xff4abcff,
    0xff67b7ff,
    0xff7eb1ff,
    0xff92a9ff,
    0xffa6a0ff,
    0xffba93ff,
    0xffd083ff,
    0xffe869ff,
    0xffff34ff,
    0xffff35ee,
    0xffff3cd3,
    0xffff3dbd,
    0xffff3aa9,
    0xffff3597,
    0xffff2e85,
    0xffff2472,
    0xffff145e,
    0xffff0046,
    0xffff0027,
    0xffff0000,
    0xff000000,
};
__device__ const int colorCount = sizeof(colors) / sizeof(colors[0]);

__device__ const float percPerColor = 1 / (float)(colorCount - 1);

__device__ int lerpColor(int low, int high, float perc) {
    int r1 = (low >> 16) & 0xFF;
    int g1 = (low >> 8) & 0xFF;
    int b1 = (low >> 0) & 0xFF;
    int r2 = (high >> 16) & 0xFF;
    int g2 = (high >> 8) & 0xFF;
    int b2 = (high >> 0) & 0xFF;
    int lr = lerp(r1, r2, perc) & 0xFF;
    int lg = lerp(g1, g2, perc) & 0xFF;
    int lb = lerp(b1, b2, perc) & 0xFF;
    return 0xFF << 24 | lr << 16 | lg << 8 | lb;
}

__device__ int getColor(float perc) {
    assert(0 <= perc && perc <= 1);
    if (perc == 1) return colors[colorCount - 1];
    int lowerIndex = floor(perc / percPerColor);
    int upperIndex = ceil(perc / percPerColor);
    assert(lowerIndex >= 0);
    assert(upperIndex < colorCount);
    if (lowerIndex == upperIndex) return colors[lowerIndex];
    assert(lowerIndex == upperIndex - 1);
    float relPerc = (perc - (lowerIndex * percPerColor)) / percPerColor;
    return lerpColor(colors[lowerIndex], colors[upperIndex], relPerc);
}

__device__ int compute(gpuType posx, gpuType posy, gpuType posz, int maxIter) {
    gpuType zx = posx;
    gpuType zy = posy;
    gpuType zz = posz;
    int i = 0;
    const int n = 8;
    gpuType sumSq = zx * zx + zy * zy + zz * zz;
    while (i < maxIter && sumSq < 4) {
        gpuType r = sqrt(sumSq);
        gpuType rn = __powf(r, n);
        gpuType phi = atan2f(zy, zx);
        gpuType theta = acosf(zz / r);
        gpuType sinTheta, cosTheta, sinPhi, cosPhi;
        __sincosf(n * theta, &sinTheta, &cosTheta);
        __sincosf(n * phi, &sinPhi, &cosPhi);
        // gpuType sinTheta = __sinf(n * theta);
        // gpuType cosTheta = __cosf(n * theta);
        // gpuType sinPhi = __sinf(n * phi);
        // gpuType cosPhi = __cosf(n * phi);
        gpuType nx = rn * sinTheta * cosPhi;
        gpuType ny = rn * sinTheta * sinPhi;
        gpuType nz = rn * cosTheta;
        zx = nx + posx;
        zy = ny + posy;
        zz = nz + posz;
        sumSq = zx * zx + zy * zy + zz * zz;
        i++;
    }
    return i;
}

__device__ void calculatePixel(gpuType *info, int id, float *paramIter, float *paramDepth) {
    gpuType Ax = info[0];
    gpuType Ay = info[1];
    gpuType Az = info[2];
    gpuType stepIx = info[3];
    gpuType stepIy = info[4];
    gpuType stepIz = info[5];
    gpuType stepJx = info[6];
    gpuType stepJy = info[7];
    gpuType stepJz = info[8];
    gpuType stepKx = info[9];
    gpuType stepKy = info[10];
    gpuType stepKz = info[11];
    int maxIter = cast(info[12], int);
    int maxDepth = cast(info[13], int);
    int _x = id % WIDTH;
    int _y = id / WIDTH;
    float totalIter = 0;
    float totalDepth = 0;
    for (int aax = 0; aax < AA_LEVEL; aax++) {
        float nx = (float)_x + (float)aax / (float)AA_LEVEL;
        for (int aay = 0; aay < AA_LEVEL; aay++) {
            float ny = (float)_y + (float)aay / (float)AA_LEVEL;
            gpuType posx = Ax - stepIx * nx - stepJx * ny;
            gpuType posy = Ay - stepIy * nx - stepJy * ny;
            gpuType posz = Az - stepIz * nx - stepJz * ny;
            int i = 0;
            int r = maxDepth;
            for (int depth = 0; depth < maxDepth; depth++) {
                i = compute(posx, posy, posz, maxIter);
                if (i == maxIter) {
                    r = depth;
                    break;
                }
                posx += stepKx;
                posy += stepKy;
                posz += stepKz;
            }
            totalIter += (float)i;
            totalDepth += (float)r;
        }
    }
    *paramIter = totalIter;
    *paramDepth = totalDepth;
}

__device__ int color(gpuType *info, float totalIter, float totalDepth) {
    int maxIter = cast(info[12], int);
    int maxDepth = cast(info[13], int);
    float scaledIter = totalIter / AA_PER_PIXEL;
    float scaledDepth = totalDepth / AA_PER_PIXEL;
    float perc = maxDepth == 1 ? scaledIter / (float)maxIter : 1 - scaledDepth / (float)maxDepth;
    return getColor(perc);
}

__global__ void mandel(gpuType *screen, int *pixels) {
    const int N = WIDTH * HEIGHT;
    for (int id = blockIdx.x * blockDim.x + threadIdx.x;
        id < N; id += blockDim.x * gridDim.x) {
        float totalIter, totalDepth;
        calculatePixel(screen, id, &totalIter, &totalDepth);
        int col = color(screen, totalIter, totalDepth);
        pixels[id] = col;
    }
}

void resetInfo(gpuType *info) {
    int defaultIter = 100;
    int defaultDepth = 100;
    // Top Left
    info[0] = -2;
    info[1] = -2;
    info[2] = 0;
    // Step X
    info[3] = 4.0 / (float)WIDTH;
    info[4] = 0;
    info[5] = 0;
    // Step Y
    info[6] = 0;
    info[7] = 4.0 / (float)HEIGHT;
    info[8] = 0;
    // Step Z
    info[9] = 0;
    info[10] = 0;
    info[11] = 0;
    info[12] = cast(defaultIter, gpuType);
    info[13] = cast(defaultDepth, gpuType);
}

Vector3 Vector3Sub(Vector3 v1, Vector3 v2)
{
    Vector3 result = { v1.x - v2.x, v1.y - v2.y, v1.z - v2.z };

    return result;
}
Vector3 Vector3Add(Vector3 v1, Vector3 v2)
{
    Vector3 result = { v1.x + v2.x, v1.y + v2.y, v1.z + v2.z };

    return result;
}

Vector3 Vector3Scale(Vector3 v, float scalar)
{
    Vector3 result = { v.x*scalar, v.y*scalar, v.z*scalar };

    return result;
}

float Vector3Length(const Vector3 v)
{
    float result = sqrtf(v.x*v.x + v.y*v.y + v.z*v.z);

    return result;
}

int main() {
    hostCalloc(pixels, WIDTH * HEIGHT, int);
    deviceCalloc(d_pixels, WIDTH * HEIGHT, int);

    InitWindow(WIDTH, HEIGHT, "ligma");

    SetTraceLogLevel(LOG_WARNING);
    SetTargetFPS(60);
    SetConfigFlags(FLAG_FULLSCREEN_MODE);

    gpuType info[14] = {0};
    resetInfo(info);
    int infoSize = sizeof(info) / sizeof(info[0]);
    deviceCalloc(d_screen, infoSize, gpuType);
    copyHostToDevice(d_screen, info, infoSize);

    invokeKernel(mandel, 4096, 512, d_screen, d_pixels);
    copyDeviceToHost(pixels, d_pixels, WIDTH * HEIGHT);
    Image img = {
        pixels,
        WIDTH,
        HEIGHT,
        1,
        PIXELFORMAT_UNCOMPRESSED_R8G8B8A8,
    };
    Texture texture = LoadTextureFromImage(img);

    Vector3 position = {0};
    Vector3 ei = {1, 0, 0};
    Vector3 ej = {0, 1, 0};
    Vector3 ek = {0, 0, 1};
    float dimScale = 0.01;
    float lastExecTime = 0;
    bool firstFrame = true;
    bool showText = true;
    while (!WindowShouldClose()) {
        float alpha = 0.01;
        bool screenUpdated = firstFrame;
        firstFrame = false;
        if (IsKeyDown(KEY_W)) {
            Vector3 ej1 = Vector3Add(Vector3Scale(ej, cos(alpha)), Vector3Scale(ek, sin(alpha)));
            Vector3 ek1 = Vector3Add(Vector3Scale(ek, cos(alpha)), Vector3Scale(ej, -sin(alpha)));
            ej = ej1;
            ek = ek1;
            screenUpdated = true;
        }
        if (IsKeyDown(KEY_S)) {
            alpha = -alpha;
            Vector3 ej1 = Vector3Add(Vector3Scale(ej, cos(alpha)), Vector3Scale(ek, sin(alpha)));
            Vector3 ek1 = Vector3Add(Vector3Scale(ek, cos(alpha)), Vector3Scale(ej, -sin(alpha)));
            ej = ej1;
            ek = ek1;
            screenUpdated = true;
        }
        if (IsKeyDown(KEY_A)) {
            Vector3 ei1 = Vector3Add(Vector3Scale(ei, cos(alpha)), Vector3Scale(ek, -sin(alpha)));
            Vector3 ek1 = Vector3Add(Vector3Scale(ek, cos(alpha)), Vector3Scale(ei, sin(alpha)));
            ei = ei1;
            ek = ek1;
            screenUpdated = true;
        }
        if (IsKeyDown(KEY_D)) {
            alpha = -alpha;
            Vector3 ei1 = Vector3Add(Vector3Scale(ei, cos(alpha)), Vector3Scale(ek, -sin(alpha)));
            Vector3 ek1 = Vector3Add(Vector3Scale(ek, cos(alpha)), Vector3Scale(ei, sin(alpha)));
            ei = ei1;
            ek = ek1;
            screenUpdated = true;
        }
        if (IsKeyDown(KEY_E)) {
            Vector3 ei1 = Vector3Add(Vector3Scale(ei, cos(alpha)), Vector3Scale(ej, sin(alpha)));
            Vector3 ej1 = Vector3Add(Vector3Scale(ej, cos(alpha)), Vector3Scale(ei, -sin(alpha)));
            ei = ei1;
            ej = ej1;
            screenUpdated = true;
        }
        if (IsKeyDown(KEY_Q)) {
            alpha = -alpha;
            Vector3 ei1 = Vector3Add(Vector3Scale(ei, cos(alpha)), Vector3Scale(ej, sin(alpha)));
            Vector3 ej1 = Vector3Add(Vector3Scale(ej, cos(alpha)), Vector3Scale(ei, -sin(alpha)));
            ei = ei1;
            ej = ej1;
            screenUpdated = true;
        }
        if (IsKeyDown(KEY_UP)) {
            position = Vector3Add(position, Vector3Scale(ek, dimScale));
            screenUpdated = true;
        }
        if (IsKeyDown(KEY_DOWN)) {
            position = Vector3Sub(position, Vector3Scale(ek, dimScale));
            screenUpdated = true;
        }
        if (IsKeyDown(KEY_Z)) {
            dimScale *= 1.01;
            screenUpdated = true;
        }
        if (IsKeyDown(KEY_X)) {
            dimScale /= 1.01;
            screenUpdated = true;
        }
        if (IsKeyDown(KEY_SPACE)) {
            position = {0};
            ei = {1, 0, 0};
            ej = {0, 1, 0};
            ek = {0, 0, 1};
            dimScale = 0.01;
            resetInfo(info);
            screenUpdated = true;
        }
        if (IsKeyDown(KEY_COMMA)) {
            int it = cast(info[12], int);
            int nw = (int)((gpuType)it / 1.01);
            if (it == nw) it--;
            else it = nw;
            if (it < 10) {
                it = 10;
            } else {
                info[12] = cast(it, gpuType);
                screenUpdated = true;
            }
        }
        if (IsKeyDown(KEY_PERIOD)) {
            int it = cast(info[12], int);
            int nw = (int)((gpuType)it * 1.01);
            if (it == nw) it++;
            else it = nw;
            info[12] = cast(it, gpuType);
            screenUpdated = true;
        }
        if (IsKeyDown(KEY_R)) {
            int depth = cast(info[13], int);
            int nw = (int)((gpuType)depth / 1.01);
            if (depth == nw) depth--;
            else depth = nw;
            if (depth < 1) {
                depth = 1;
            } else {
                info[13] = cast(depth, gpuType);
                screenUpdated = true;
            }
        }
        if (IsKeyDown(KEY_T)) {
            int depth = cast(info[13], int);
            int nw = (int)((gpuType)depth * 1.01);
            if (depth == nw) depth++;
            else depth = nw;
            info[13] = cast(depth, gpuType);
            screenUpdated = true;
        }
        if (IsKeyDown(KEY_Y)) {
            int depth = 1;
            info[13] = cast(depth, gpuType);
            screenUpdated = true;
        }
        if (IsKeyReleased(KEY_TAB)) showText = !showText;
        if (screenUpdated) {
            // Step X
            info[3] = ei.x * dimScale;
            info[4] = ei.y * dimScale;
            info[5] = ei.z * dimScale;
            // Step Y
            info[6] = ej.x * dimScale;
            info[7] = ej.y * dimScale;
            info[8] = ej.z * dimScale;
            // Step Z
            info[9] = ek.x * dimScale;
            info[10] = ek.y * dimScale;
            info[11] = ek.z * dimScale;
            // Top Left
            info[0] = position.x + info[3] * WIDTH / 2 + info[6] * HEIGHT / 2;
            info[1] = position.y + info[4] * WIDTH / 2 + info[7] * HEIGHT / 2;
            info[2] = position.z + info[5] * WIDTH / 2 + info[8] * HEIGHT / 2;
            copyHostToDevice(d_screen, info, infoSize);
            long long int ns;
            TIME_AND_BIND(ns,
                invokeKernel(mandel, 4096, 512, d_screen, d_pixels);
                ENSURE(hipStreamSynchronize(0), "Could not synchronize Stream")
            );
            lastExecTime = (float)ns / 1'000'000;
            copyDeviceToHost(pixels, d_pixels, WIDTH * HEIGHT);
            UnloadTexture(texture);
            texture = LoadTextureFromImage(img);
        }
        BeginDrawing();
        ClearBackground(BLACK);
        DrawTexture(texture, 0, 0, WHITE);
        if (showText) {
            DrawText(TextFormat("Center: (%.12f, %.12f, %.12f)", position.x, position.y, position.z), 50, 50, 50, BLUE);
            DrawText(TextFormat("ei: (%.12f, %.12f, %.12f, %.12f)", ei.x, ei.y, ei.z, Vector3Length(ei)), 50, 100, 50, BLUE);
            DrawText(TextFormat("ej: (%.12f, %.12f, %.12f, %.12f)", ej.x, ej.y, ej.z, Vector3Length(ej)), 50, 150, 50, BLUE);
            DrawText(TextFormat("ek: (%.12f, %.12f, %.12f, %.12f)", ek.x, ek.y, ek.z, Vector3Length(ek)), 50, 200, 50, BLUE);
            DrawText(TextFormat("Kernel took: %.6f ms", lastExecTime), 50, 250, 50, BLUE);
            DrawText(TextFormat("Iterations: %d", cast(info[12], int)), 50, 300, 50, BLUE);
            DrawText(TextFormat("Depth: %d", cast(info[13], int)), 50, 350, 50, BLUE);
        }
        EndDrawing();
    }

    freeHost();
    freeDevice();
    return 0;
}
